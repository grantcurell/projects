#include "hip/hip_runtime.h"
// gpupcapgrep.cu
// CUDA-accelerated PCAP string search using Boyer–Moore–Horspool.
// Searches entire frames; handles many small packets and large packets efficiently.
// Build: nvcc -O3 -std=c++17 -Xcompiler -fopenmp -lpcap -o gpupcapgrep gpupcapgrep.cu

#include <pcap/pcap.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <cstring>
#include <string>
#include <vector>
#include <iostream>
#include <stdexcept>
#include <algorithm>

#define CUDA_CHECK(call) do { hipError_t e = (call); if (e != hipSuccess) { \
  fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); exit(1);} } while(0)

struct Pattern {
  std::string bytes;
  uint8_t badchar[256]; // BMH bad-character shift table
};

struct Match {
  uint32_t packet_id;
  uint32_t offset;
  uint16_t pattern_id;
};

static void make_bmh(const std::string &pat, uint8_t table[256]) {
  const int m = (int)pat.size();
  for (int i=0;i<256;i++) table[i] = (uint8_t)m; // default shift = m
  for (int i=0;i<m-1;i++) table[(uint8_t)pat[i]] = (uint8_t)(m-1-i);
}

// Device constant memory for one pattern at a time (fast access)
__constant__ uint8_t d_pat[512];        // up to 512-byte pattern (tune as needed)
__constant__ int d_pat_len;
__constant__ uint8_t d_badchar[256];

struct PacketView { const uint8_t* base; uint32_t len; };

// Global match buffer bookkeeping
struct GMatches {
  Match* entries;
  unsigned long long* counter;
  unsigned long long capacity;
};

__device__ __forceinline__ bool bmh_find_once(const uint8_t* buf, int n) {
  // Returns true if at least one match exists (used for early-exit variants if needed)
  int m = d_pat_len;
  if (m==0 || n<m) return false;
  int i = 0;
  while (i <= n - m) {
    uint8_t last = buf[i + m - 1];
    int j = m - 1;
    // Compare backwards
    while (j >= 0 && d_pat[j] == buf[i + j]) { --j; }
    if (j < 0) return true;
    i += d_badchar[last];
  }
  return false;
}

__device__ __forceinline__ void bmh_scan_emit(const uint8_t* buf, int n, uint32_t pkt_id, uint16_t pat_id, GMatches g) {
  int m = d_pat_len;
  if (m==0 || n<m) return;
  // stride by blockDim.x * gridDim.x (but we'll use per-block stride below)
  int start = threadIdx.x;
  int step  = blockDim.x;
  // We implement BMH per-thread by scanning candidate windows at positions that this thread owns
  // To preserve correctness, we can't just skip by d_badchar because that's per window; implement a hybrid:
  // Each thread advances 'i' locally with BMH skip logic, starting at its thread-local i.
  for (int i = start; i <= n - m; ) {
    const uint8_t last = buf[i + m - 1];
    int j = m - 1;
    while (j >= 0 && d_pat[j] == buf[i + j]) { --j; }
    if (j < 0) {
      // Emit match
      unsigned long long idx = atomicAdd(g.counter, 1ULL);
      if (idx < g.capacity) {
        g.entries[idx] = Match{pkt_id, (uint32_t)i, pat_id};
      }
      i += m; // on match, advance by m (standard BMH choice)
    } else {
      int shift = d_badchar[last];
      if (shift < 1) shift = 1;
      i += shift;
    }
    // Ensure each thread doesn't starve others; no sync needed
  }
}

// Tunables
constexpr int BLOCK_SIZE = 256;
constexpr int LARGE_PKT_THRESHOLD = 2048;   // bytes
constexpr int TILE_BYTES = 8192;            // shared-memory tile size (<= 48-96KB SMEM budgets across GPUs)
constexpr int OVERLAP_MAX = 511;            // must be >= max pattern length-1 (we set pattern <=512)

__global__ void kernel_small_packets(const uint8_t* __restrict__ bigbuf,
                                     const uint32_t* __restrict__ offsets,
                                     const uint32_t* __restrict__ lengths,
                                     int num_packets,
                                     uint16_t pattern_id,
                                     GMatches g)
{
  int pkt = blockIdx.x;
  if (pkt >= num_packets) return;
  int m = d_pat_len;
  if (m <= 0) return;

  const uint8_t* pkt_ptr = bigbuf + offsets[pkt];
  int n = (int)lengths[pkt];
  if (n < m) return;
  bmh_scan_emit(pkt_ptr, n, pkt, pattern_id, g);
}

__global__ void kernel_large_packets(const uint8_t* __restrict__ bigbuf,
                                     const uint32_t* __restrict__ offsets,
                                     const uint32_t* __restrict__ lengths,
                                     const int* __restrict__ large_indices,
                                     int num_large,
                                     uint16_t pattern_id,
                                     GMatches g)
{
  extern __shared__ uint8_t smem[]; // TILE_BYTES + OVERLAP_MAX
  int list_idx = blockIdx.x;
  if (list_idx >= num_large) return;

  int pkt = large_indices[list_idx];
  int m = d_pat_len;
  if (m <= 0) return;

  const uint8_t* pkt_ptr = bigbuf + offsets[pkt];
  int n = (int)lengths[pkt];
  if (n < m) return;

  // Process packet in tiles with m-1 overlap
  for (int base = 0; base < n; base += TILE_BYTES) {
    int tile_len = min(TILE_BYTES, n - base);
    int overlap = (m > 0) ? (m - 1) : 0;
    int copy_len = tile_len + overlap;
    if (base + copy_len > n) copy_len = n - base;

    // Cooperative load into shared memory
    for (int i = threadIdx.x; i < copy_len; i += blockDim.x) {
      smem[i] = pkt_ptr[base + i];
    }
    __syncthreads();

    // Each thread scans the tile (excluding the final overlap-only region)
    int scan_len = tile_len; // valid starts within [0, tile_len - m]
    const uint8_t* buf = smem;
    int max_start = scan_len - m;
    if (max_start >= 0) {
      int i = threadIdx.x;
      while (i <= max_start) {
        uint8_t last = buf[i + m - 1];
        int j = m - 1;
        while (j >= 0 && d_pat[j] == buf[i + j]) { --j; }
        if (j < 0) {
          unsigned long long idx = atomicAdd(g.counter, 1ULL);
          if (idx < g.capacity) {
            g.entries[idx] = Match{(uint32_t)pkt, (uint32_t)(base + i), pattern_id};
          }
          i += m;
        } else {
          int shift = d_badchar[last];
          if (shift < 1) shift = 1;
          i += shift;
        }
      }
    }
    __syncthreads();
  }
}

// Simple host-side PCAP loader: concatenates all packet bytes into one big buffer
struct PcapData {
  std::vector<uint8_t> bigbuf;
  std::vector<uint32_t> offsets;
  std::vector<uint32_t> lengths;
};

static PcapData load_pcap(const char* path) {
  char errbuf[PCAP_ERRBUF_SIZE];
  pcap_t* p = pcap_open_offline(path, errbuf);
  if (!p) throw std::runtime_error(std::string("pcap_open_offline: ") + errbuf);

  PcapData out;
  out.bigbuf.reserve(1<<26); // pre-reserve 64MB
  const u_char* pkt;
  struct pcap_pkthdr hdr;

  uint64_t total = 0;
  while ((pkt = pcap_next(p, &hdr)) != nullptr) {
    out.offsets.push_back((uint32_t)out.bigbuf.size());
    out.lengths.push_back((uint32_t)hdr.caplen);
    out.bigbuf.insert(out.bigbuf.end(), pkt, pkt + hdr.caplen);
    total += hdr.caplen;
  }
  pcap_close(p);
  // Ensure at least one packet
  if (out.offsets.empty()) throw std::runtime_error("No packets in PCAP.");
  return out;
}

static void usage(const char* argv0) {
  std::cerr << "Usage: " << argv0 << " file.pcap -s <string> [-s <string> ...]\n"
            << "Strings can include C-style hex escapes like \\x00.\n";
}

static std::string unescape(const std::string &in) {
  std::string out;
  out.reserve(in.size());
  for (size_t i=0;i<in.size();) {
    if (in[i]=='\\' && i+1<in.size()) {
      if (in[i+1]=='x' && i+3<in.size()) {
        auto hex = in.substr(i+2,2);
        uint8_t v = (uint8_t)strtoul(hex.c_str(), nullptr, 16);
        out.push_back((char)v);
        i+=4;
      } else {
        char c = in[i+1];
        if (c=='n') out.push_back('\n');
        else if (c=='r') out.push_back('\r');
        else if (c=='t') out.push_back('\t');
        else out.push_back(c);
        i+=2;
      }
    } else {
      out.push_back(in[i++]);
    }
  }
  return out;
}

int main(int argc, char** argv) {
  if (argc < 3) { usage(argv[0]); return 1; }
  const char* pcap_path = argv[1];
  std::vector<Pattern> patterns;

  for (int i=2;i<argc;i++) {
    if (std::string(argv[i]) == "-s") {
      if (i+1>=argc) { usage(argv[0]); return 1; }
      std::string raw = argv[++i];
      std::string un = unescape(raw);
      Pattern p; p.bytes = std::move(un);
      if (p.bytes.empty()) { std::cerr << "Empty pattern ignored\n"; continue; }
      if (p.bytes.size() > 512) { std::cerr << "Pattern too long (max 512 bytes)\n"; return 1; }
      make_bmh(p.bytes, p.badchar);
      patterns.push_back(std::move(p));
    } else {
      std::cerr << "Unknown arg: " << argv[i] << "\n";
      usage(argv[0]);
      return 1;
    }
  }
  if (patterns.empty()) { std::cerr << "No patterns provided.\n"; return 1; }

  // Load PCAP
  PcapData pd;
  try {
    pd = load_pcap(pcap_path);
  } catch (const std::exception &e) {
    std::cerr << "Error: " << e.what() << "\n";
    return 1;
  }

  const int num_packets = (int)pd.offsets.size();

  // Host-side: compute list of large packets indices for the large-kernel path
  std::vector<int> large_indices;
  large_indices.reserve(num_packets/4);
  for (int i=0;i<num_packets;i++) {
    if (pd.lengths[i] >= LARGE_PKT_THRESHOLD) large_indices.push_back(i);
  }

  // Device buffers
  uint8_t* d_bigbuf = nullptr;
  uint32_t *d_offsets=nullptr, *d_lengths=nullptr;
  int* d_large_indices = nullptr;

  CUDA_CHECK(hipMalloc((void**)&d_bigbuf, pd.bigbuf.size()));
  CUDA_CHECK(hipMemcpy(d_bigbuf, pd.bigbuf.data(), pd.bigbuf.size(), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMalloc((void**)&d_offsets, num_packets*sizeof(uint32_t)));
  CUDA_CHECK(hipMalloc((void**)&d_lengths, num_packets*sizeof(uint32_t)));
  CUDA_CHECK(hipMemcpy(d_offsets, pd.offsets.data(), num_packets*sizeof(uint32_t), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_lengths, pd.lengths.data(), num_packets*sizeof(uint32_t), hipMemcpyHostToDevice));

  if (!large_indices.empty()) {
    CUDA_CHECK(hipMalloc((void**)&d_large_indices, large_indices.size()*sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_large_indices, large_indices.data(), large_indices.size()*sizeof(int), hipMemcpyHostToDevice));
  }

  // Global matches buffer (pre-allocate generous capacity)
  const unsigned long long max_matches = std::max<unsigned long long>(num_packets * 8ULL, 1ULL << 20); // heuristic
  Match* d_matches = nullptr;
  unsigned long long* d_count = nullptr;
  CUDA_CHECK(hipMalloc((void**)&d_matches, max_matches * sizeof(Match)));
  CUDA_CHECK(hipMalloc((void**)&d_count, sizeof(unsigned long long)));

  // For each pattern: upload to constant memory and launch kernels
  std::vector<Match> h_matches; h_matches.reserve(1024);
  for (size_t pidx=0; pidx<patterns.size(); ++pidx) {
    const Pattern &pat = patterns[pidx];

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_pat), pat.bytes.data(), pat.bytes.size(), 0, hipMemcpyHostToDevice));
    int h_m = (int)pat.bytes.size();
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_pat_len), &h_m, sizeof(int), 0, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_badchar), pat.badchar, 256, 0, hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemset(d_count, 0, sizeof(unsigned long long)));

    // Launch small packets kernel
    int small_packets = num_packets - (int)large_indices.size();
    if (num_packets > 0) {
      dim3 blocks_small(num_packets); // one block per packet; small packets exit quickly
      kernel_small_packets<<<blocks_small, BLOCK_SIZE>>>(
        d_bigbuf, d_offsets, d_lengths, num_packets,
        (uint16_t)pidx,
        GMatches{d_matches, d_count, max_matches});
    }

    // Launch large packets kernel with shared-memory tiling
    if (!large_indices.empty()) {
      dim3 blocks_large((int)large_indices.size());
      size_t smem_bytes = TILE_BYTES + OVERLAP_MAX;
      kernel_large_packets<<<blocks_large, BLOCK_SIZE, smem_bytes>>>(
        d_bigbuf, d_offsets, d_lengths, d_large_indices, (int)large_indices.size(),
        (uint16_t)pidx,
        GMatches{d_matches, d_count, max_matches});
    }

    CUDA_CHECK(hipDeviceSynchronize());

    // Copy matches back
    unsigned long long count = 0;
    CUDA_CHECK(hipMemcpy(&count, d_count, sizeof(unsigned long long), hipMemcpyDeviceToHost));
    if (count > max_matches) count = max_matches;

    std::vector<Match> temp(count);
    if (count) {
      CUDA_CHECK(hipMemcpy(temp.data(), d_matches, count*sizeof(Match), hipMemcpyDeviceToHost));
      // Accumulate
      h_matches.insert(h_matches.end(), temp.begin(), temp.end());
    }
  }

  // Cleanup device
  hipFree(d_bigbuf); hipFree(d_offsets); hipFree(d_lengths);
  if (d_large_indices) hipFree(d_large_indices);
  hipFree(d_matches); hipFree(d_count);

  // Sort matches for stable output: by packet, then offset, then pattern
  std::sort(h_matches.begin(), h_matches.end(), [](const Match&a, const Match&b){
    if (a.packet_id != b.packet_id) return a.packet_id < b.packet_id;
    if (a.offset != b.offset) return a.offset < b.offset;
    return a.pattern_id < b.pattern_id;
  });

  // Print results
  for (const auto &m : h_matches) {
    std::cout << "packet=" << m.packet_id
              << " offset=" << m.offset
              << " pattern=" << m.pattern_id
              << "\n";
  }

  return 0;
}
